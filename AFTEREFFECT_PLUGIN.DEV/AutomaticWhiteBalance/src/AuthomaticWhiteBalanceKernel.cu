#include "hip/hip_runtime.h"
#include "ImageLabCUDA.hpp"
#include "AuthomaticWhiteBalanceGPU.hpp"
#include "ColorTransformMatrix.hpp"
#include "FastAriphmetics.hpp"
#include "AlgCorrectionMatrix.hpp"
#include <hip/hip_runtime.h>
#include <math.h>


float4* RESTRICT gpuImage[2]{ nullptr };

//////////////////////// PURE DEVICE CODE ///////////////////////////////////////////
inline __device__ float4 HalfToFloat4(Pixel16 in) noexcept
{
    return make_float4(__half2float(in.x), __half2float(in.y), __half2float(in.z), __half2float(in.w));
}

inline __device__ Pixel16 FloatToHalf4(float4 in) noexcept
{
    Pixel16 v;
    v.x = __float2half_rn(in.x); v.y = __float2half_rn(in.y); v.z = __float2half_rn(in.z); v.w = __float2half_rn(in.w);
    return v;
}


CUDA_KERNEL_CALL
void AuthomaticWhiteBalance_CUDA
(
    float* inBuf,
    float* outBuf,
    int destPitch,
    int srcPitch,
    int	is16f,
    int width,
    int height,
    const eILLUMINATE illuminant,
    const eChromaticAdaptation chroma,
    const eCOLOR_SPACE color_space,
    const float gray_threshold,
    unsigned int iter_cnt
)
{
    float4* gpuTmpImage = nullptr;
    float4* src = nullptr;
    float4* dst = nullptr;
    int srcIdx = 0, dstIdx = 1;
    int inPitch = 0, outPitch = 0;

    dim3 blockDim(32, 32, 1);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y, 1);

    // allocate memory for intermediate processing results
    const unsigned int blocksNumber = FastCompute::Min(2u, (iter_cnt - 1u));
    if (blocksNumber > 0)
    {
        const unsigned int frameSize = static_cast<unsigned int>(width) * static_cast<unsigned int>(height);
        if (hipSuccess == hipMalloc(reinterpret_cast<void**>(&gpuTmpImage), blocksNumber * frameSize * sizeof(float4)))
        {
            gpuImage[0] = reinterpret_cast<float4* RESTRICT>(gpuTmpImage);
            gpuImage[1] = (2u == blocksNumber ? gpuImage[0] + frameSize : nullptr);
        }
    } // if (blocksNumber > 0)
    
    // MAIN PROC LOOP
    for (unsigned int i = 0u; i < iter_cnt; i++)
    {

    } // for (unsigned int i = 0u; i < iter_cnt; i++)

    // Free/Cleanup resources before exit
    if (nullptr != gpuTmpImage)
    {
        hipFree(gpuTmpImage);
        gpuTmpImage = gpuImage[0] = gpuImage[1] = nullptr;
    }

   return;
}