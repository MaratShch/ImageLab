#include "hip/hip_runtime.h"
#include "SepiaColorGPU.hpp"
#include "SepiaMatrix.hpp"
#include "FastAriphmetics.hpp"


__constant__ float gpuSepiaMatrix[9];


inline __device__ float4 HalfToFloat4 (Pixel16 in)
{
	return make_float4 (__half2float(in.x), __half2float(in.y), __half2float(in.z), __half2float(in.w));
}

inline __device__ Pixel16 FloatToHalf4(float4 in)
{
	Pixel16 v;
	v.x = __float2half_rn(in.x); v.y = __float2half_rn(in.y); v.z = __float2half_rn(in.z); v.w = __float2half_rn(in.w);
	return v;
}


__global__ void kSepiaColorCUDA
(
	float4* inImg,
	float4* outImg,
	const int destPitch,
	const int in16f,
	const int inWidth,
	const int inHeight
)
{
	float4 in;
	float4 out;

	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x >= inWidth || y >= inHeight) return;

	if (in16f) {
		Pixel16*  in16 = (Pixel16*)outImg;
		out = HalfToFloat4(in16[y *  destPitch + x]);
	}
	else {
		out = outImg[y *  destPitch + x];
	}


	return;
}


CUDA_KERNEL_CALL
bool SepiaColorLoadMatrix_CUDA(void)
{
	/* SepiaMatrix array is defined in "SepiaMatrix.hpp" include file */
	constexpr size_t loadSize = sizeof(SepiaMatrix);
	const hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(gpuSepiaMatrix), SepiaMatrix, loadSize);

#ifdef _DEBUG
	float dbg_gpuSepiaMatrix[9] = {};
	hipMemcpyFromSymbol(dbg_gpuSepiaMatrix, HIP_SYMBOL(gpuSepiaMatrix), loadSize);
	for (int i = 0; i < 9; i++)
	{
		if (dbg_gpuSepiaMatrix[i] != SepiaMatrix[i])
			return false;
	}
	return true;
#else
	return (hipSuccess == err) ? true : false;
#endif
}


CUDA_KERNEL_CALL
void SepiaColor_CUDA
(
	float* inBuf,
	float* outBuf,
	int destPitch,
	int	is16f,
	int width,
	int height
)
{
	dim3 blockDim(16, 32, 1);
	dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y, 1);

	kSepiaColorCUDA <<< gridDim, blockDim, 0 >>> ((float4*)inBuf, (float4*)outBuf, destPitch, is16f, width, height);

	hipDeviceSynchronize();

	return;
}
